#include "hip/hip_runtime.h"
//
// Created by sivecano on 02/05/2021.
//
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "bufferinterface.cuh"
#include <stdexcept>
#include "stdio.h"
#include <vector>
#include <utility>
#include <string>

std::vector<std::pair<unsigned int, hipGraphicsResource_t>> buffers;

__device__ inline unsigned int bindex(unsigned int x, unsigned int y, unsigned int z)
{
    return x + 16 * z + 256 * y;
}
//TODO: implement using the remaining two bytes as cross so we can have flowers and shit
/**
 * calculate which faces within the chunk need to be drawn
 *    _______
 *   /______/|
 *  |      | |
 *  |      | |
 *  |______|/
 *  /\
 *  | y
 *  -->x
 *  bitmapping: (from least to most significant bit)
 *
 *  0: bottom
 *  1: right
 *  2: back
 *  3: left
 *  4: front
 *  5: top
 *
 *  we still have to remaining bits that currently serve no use
 *
 *
 * @param buffer data consisting of blocktypes and facemask space
 */
__global__ void calculate_cube_facemask(uint8_t* data)
{
    uint8_t out = 0x3f;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;  // location
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0) if (data[bindex(x - 1, y, z)] != 0) out &= ~16;      // left
    if (y > 0) if (data[bindex(x, y - 1, z)] != 0) out &= ~1;      // bottom
    if (z > 0) if (data[bindex(x, y, z - 1)] != 0) out &= ~2;      // front // what happenes if u acess data out of chunk?????? nothing... you just access another part of the array
// oh ic kk
//oh, you mean outside chunk? that's why we have the first "if". we don't access that
    if (x < 15) if (data[bindex(x + 1, y, z)] != 0) out &= ~4;     // right
    if (y < 15) if (data[bindex(x, y + 1, z)] != 0) out &= ~32;    // top
    if (z < 15) if (data[bindex(x, y, z + 1)] != 0) out &= ~8;    // back

    //printf("x: %i, y: %i, z: %i\n", x, y, z);

    if ((x == 16) && (y == 0) && (z == 0)) {
        printf(" curr : %i, x + 1: %i, y + 1: %i, z + 1: %i", data[bindex(x, y, z)], data[bindex(x + 1, y, z)], data[bindex(x, y+1, z)], data[bindex(x, y, z+1)]);
        printf(", x - 1: %i\t", data[bindex(x - 1, y, z)]);
        printf("out: %i\n", out);
    }

    data[4096 + bindex(x, y, z)] = out;
}


void cudainit() {

    printf("error: %s\n", hipGetErrorString(hipGetLastError()));
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("hipSetDevice failed!  ") + std::string(hipGetErrorString(err)));
    }
}

void register_buffer(unsigned int buffer)
{
    hipGraphicsResource_t resource;
    hipGraphicsGLRegisterBuffer(&resource, buffer, hipGraphicsRegisterFlagsNone);
    hipGraphicsMapResources(1, &resource);
    buffers.push_back(std::pair<unsigned int, hipGraphicsResource_t>(buffer, resource));
}

void unregister_buffer(unsigned int buffer)
{
    hipGraphicsResource_t resource;
    for (auto el = buffers.begin(); el != buffers.end(); el++)
        if ((*el).first == buffer) {
            resource = (*el).second;
            hipGraphicsUnmapResources(1, &resource);
            hipGraphicsUnregisterResource(resource);
            el = buffers.erase(el, el + 1);
        }
}

hipGraphicsResource_t get_resource(unsigned int buffer)
{
    for (auto el = buffers.begin(); el != buffers.end(); el++)
        if ((*el).first == buffer)
             return (*el).second;

    return NULL;
}


void cube_facemask(unsigned int blockbuffer)
{
    void* pointer;
    size_t size;
    hipGraphicsResourceGetMappedPointer(&pointer, &size , get_resource(blockbuffer));
    hipError_t err;

    dim3 block;
    block.x = 2;
    block.y = 2;
    block.z = 2;

    dim3 thread;
    thread.x = 8;
    thread.y = 8;
    thread.z = 8;

    calculate_cube_facemask<<<block, thread>>>((uint8_t*)pointer);

    err = hipDeviceSynchronize();
    //printf("return: %s", hipGetErrorName(err));
    if (err != hipSuccess)
        printf("we fucked up majorly in facemask kernel: %s\n", hipGetErrorString(err));
}

//void cube_facemask(Chunksystem world, glm::ivec3 chunkpos) {

//}
