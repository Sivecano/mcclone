#include "hip/hip_runtime.h"
//
// Created by sivecano on 02/05/2021.
//
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "kernels.cuh"
#include <stdexcept>
#include "stdio.h"

hipGraphicsResource_t blockbuffer;

__device__ inline unsigned int bindex(unsigned int x, unsigned int y, unsigned int z)
{
    return x + 16 * y + 256 * z;
}

/**
 * calculate which faces within the chunk need to be drawn
 *    _______
 *   /______/|
 *  |      | |
 *  |      | |
 *  |______|/
 *
 *  bitmapping: (from least to most significant bit)
 *
 *  0: bottom
 *  1: right
 *  2: back
 *  3: left
 *  4: front
 *  5: top
 *
 *  we still have to remaining bits that currently serve no use
 *
 *
 * @param buffer data consisting of blocktypes and facemask space
 */
__global__ void calculate_cube_facemask(uint8_t* data)
{
    uint8_t out = 0xff;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;  // location
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0) if (data[bindex(x - 1, y, z)] != 0) out &= ~8;      // left
    if (y > 0) if (data[bindex(x, y - 1, z)] != 0) out &= ~1;      // bottom
    if (z > 0) if (data[bindex(x, y, z - 1)] != 0) out &= ~16;      // front // what happenes if u acess data out of chunk?????? nothing... you just access another part of the array
// oh ic kk
//oh, you mean outside chunk? that's why we have the first "if". we don't access that
    if (x < 15) if (data[bindex(x + 1, y, z)] != 0) out &= ~2;     // right
    if (y < 15) if (data[bindex(x, y + 1, z)] != 0) out &= ~32;    // top
    if (z < 15) if (data[bindex(x, y, z + 1)] != 0) out &= ~4;    // back

    //printf("x: %i, y: %i, z: %i\n", x, y, z);

    if ((x == 16) && (y == 0) && (z == 0)) {
        printf(" curr : %i, x + 1: %i, y + 1: %i, z + 1: %i", data[bindex(x, y, z)], data[bindex(x + 1, y, z)], data[bindex(x, y+1, z)], data[bindex(x, y, z+1)]);
        printf(", x - 1: %i\t", data[bindex(x - 1, y, z)]);
        printf("out: %i\n", out);
    }

    data[4096 + bindex(x, y, z)] = out;
}


void cudainit() {
    if (hipSetDevice(0) != hipSuccess) {
        throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
}

hipGraphicsResource_t register_buffer(int buffer)
{
    hipGraphicsResource_t resource;
    hipGraphicsGLRegisterBuffer(&resource, buffer, hipGraphicsRegisterFlagsNone);
    hipGraphicsMapResources(1, &resource);
    return resource;
}

void unregister_buffer(hipGraphicsResource_t resource)
{
    hipGraphicsUnmapResources(1, &resource);
    hipGraphicsUnregisterResource(resource);
}

void register_blockbuffer(int buffer) {
    blockbuffer = register_buffer(buffer);
}

void unregister_blockbuffer()
{
    unregister_buffer(blockbuffer);
}

void cube_facemask()
{
    void* pointer;
    size_t size;
    hipGraphicsResourceGetMappedPointer(&pointer, &size , blockbuffer);
    hipError_t err;

    dim3 block;
    block.x = 2;
    block.y = 2;
    block.z = 2;

    dim3 thread;
    thread.x = 8;
    thread.y = 8;
    thread.z = 8;

    calculate_cube_facemask<<<block, thread>>>((uint8_t*)pointer);

    err = hipDeviceSynchronize();
    //printf("return: %s", hipGetErrorName(err));
    if (err != hipSuccess)
        printf("we fucked up majorly: %s\n", hipGetErrorString(err));
}
