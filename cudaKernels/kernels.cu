#include "hip/hip_runtime.h"
//
// Created by sivecano on 02/05/2021.
//
#include "cuda_gl_interop.h"
#include "kernels.cuh"
/**
 * calculate which faces within the chunk need to be drawn
 *    _______
 *   /______/|
 *  |      | |
 *  |      | |
 *  |______|/
 *
 *  bitmapping:
 *
 *  0: bottom
 *  1: right
 *  2: back
 *  3: left
 *  4: front
 *  5: top
 *
 *  we still have to remaining bits that currently serve no use
 *
 *
 * @param buffer data consisting of blocktypes and facemask space
 */

hipGraphicsResource_t blockbuffer;

__global__ void calculate_cube_facemask(uint8_t* data)
{
    uint8_t out = 0xff;
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    uint z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x > 0) if (data[(x - 1) + 16 * y + 256*z] == 0) out &= ~8;     // left
    if (y > 0) if (data[x + 16 * (y - 1) + 256*z] == 0) out &= ~1;   // bottom
    if (z > 0) if (data[x + 16 * y + 256*(z - 1)] == 0) out &= ~16;  //front

    if (x < 15) if (data[(x + 1) + 16 * y + 256*z] == 0) out &= ~2;    // right
    if (y < 15) if (data[x + 16 * (y + 1) + 256*z] == 0) out &= ~32; // top
    if (z < 15) if (data[x + 16 * y + 256*(z + 1)] == 0) out &= ~4;  // back

    data[4096 + x + 16 * y + 256 * z] = out;
}


void cudainit() {
    if (hipSetDevice(0) != hipSuccess) {
        throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
}

hipGraphicsResource_t register_buffer(uint buffer)
{
    hipGraphicsResource_t resource;
    hipGraphicsGLRegisterBuffer(&resource, buffer, hipGraphicsRegisterFlagsNone);
    hipGraphicsMapResources(1, &resource);
    return resource;
}

void unregister_buffer(hipGraphicsResource_t resource)
{
    hipGraphicsUnmapResources(1, &resource);
    hipGraphicsUnregisterResource(resource);
}

void register_blockbuffer(GLuint buffer) {
    blockbuffer = register_buffer(buffer);
}

void unregister_blockbuffer()
{
    unregister_buffer(blockbuffer);
}

void cube_facemask()
{
    void* pointer;
    size_t size;
    hipGraphicsResourceGetMappedPointer(&pointer, &size , blockbuffer);

    dim3 block = {2,2,2};
    dim3 thread = {8,8,8};
    calculate_cube_facemask<<<block, thread>>>((uint8_t*)pointer);
    hipDeviceSynchronize();
}
