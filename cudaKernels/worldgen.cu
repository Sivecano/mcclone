#include "hip/hip_runtime.h"
//
// Created by sivecano on 04/05/2021.
//

#include "worldgen.cuh"
#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ inline unsigned int bindex(unsigned int x, unsigned int y, unsigned int z)
{
    return x + 16 * z + 256 * y;
}

__global__ void generate(uint8_t* blockids, int64_t chunkx, int64_t chunky, int64_t chunkz)
{
    int64_t x = blockIdx.x * blockDim.x + threadIdx.x;  // location
    int64_t y = blockIdx.y * blockDim.y + threadIdx.y;
    int64_t z = blockIdx.z * blockDim.z + threadIdx.z;

    blockids[bindex(x, y, z)] = (((sin(0.1 * (x + chunkx)) + 1) * abs(x - 8)  + (cos(0.1 * (z + chunkz)) + 1) * abs(z - 8) < (y + chunky) / 2)) ? 0 : 7;
}


void generate_chunkids(uint8_t* chunkids, glm::ivec3 chunkpos)
{
    chunkpos *= 16;

    hipError_t err;

    dim3 block;
    block.x = 2;
    block.y = 2;
    block.z = 2;

    dim3 thread;
    thread.x = 8;
    thread.y = 8;
    thread.z = 8;

    uint8_t* devptr;

    err = hipMalloc(&devptr, 4096);

    if (err != hipSuccess)
        printf("we fucked up in allocating gpu memory: %s\n", hipGetErrorString(err));


    generate<<<block, thread>>>(devptr, chunkpos.x, chunkpos.y, chunkpos.z);

    err = hipDeviceSynchronize();
    //printf("return: %s", hipGetErrorName(err));
    if (err != hipSuccess)
        printf("we fucked up in generating chunk at %i, %i, %i: %s\n", chunkpos.x, chunkpos.y, chunkpos.z, hipGetErrorString(err));

    err = hipMemcpy(chunkids, devptr, 4096, hipMemcpyDeviceToHost);
    hipFree(devptr);
}