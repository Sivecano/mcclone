#include "hip/hip_runtime.h"
//
// Created by sivecano on 04/05/2021.
//

#include "worldgen.cuh"
#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ inline unsigned int bindex(unsigned int x, unsigned int y, unsigned int z)
{
    return x + 16 * z + 256 * y;
}

__global__ void generate(uint8_t* blockids, int chunkx, int chunky, int chunkz)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;  // location
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

    blockids[bindex(x, y, z)] = ((2*(x + z - 16) < (y + chunky)) || (2*(16 - x + z) < (y + chunky))) ? 0 : 7;
}


void generate_chunkids(uint8_t* chunkids, glm::ivec3 chunkpos)
{
    chunkpos *= 16;

    hipError_t err;

    dim3 block;
    block.x = 2;
    block.y = 2;
    block.z = 2;

    dim3 thread;
    thread.x = 8;
    thread.y = 8;
    thread.z = 8;

    uint8_t* devptr;

    err = hipMalloc(&devptr, 4096);

    if (err != hipSuccess)
        printf("we fucked up in allocating gpu memory: %s\n", hipGetErrorString(err));


    generate<<<block, thread>>>(devptr, chunkpos.x, chunkpos.y, chunkpos.z);

    err = hipDeviceSynchronize();
    //printf("return: %s", hipGetErrorName(err));
    if (err != hipSuccess)
        printf("we fucked up in generating chunk at %i, %i, %i: %s\n", chunkpos.x, chunkpos.y, chunkpos.z, hipGetErrorString(err));

    err = hipMemcpy(chunkids, devptr, 4096, hipMemcpyDeviceToHost);
    hipFree(devptr);
}